#ifndef BLAS_CPP
#define BLAS_CPP

#include "blas.h"


void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, float *alpha, float *A, int lda, float *B, int ldb, float *beta, float *C, int ldc) {
  hipblasSgemm(handle, transa, transb, m, n, k, (const float *)alpha, (const float *)A, lda, (const float *)B, ldb, (const float *)beta, C, ldc);
}

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, double *alpha, double *A, int lda, double *B, int ldb, double *beta, double *C, int ldc) {
  hipblasDgemm(handle, transa, transb, m, n, k, (const double *)alpha, (const double *)A, lda, (const double *)B, ldb, (const double *)beta, C, ldc);
}

// void gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, float *alpha, float *A, int lda, float *x, int incx, float *beta, float *y, int incy) {
// 	hipblasSgemv(handle, trans, m, n, (const float *)alpha, (const float *)A, lda, (const float *)x, incx, (const float *)beta, y, incy);
// }

// void gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, double *alpha, double *A, int lda, double *x, int incx, double *beta, double *y, int incy) {
// 	hipblasDgemv(handle, trans, m, n, (const double *)alpha, (const double *)A, lda, (const double *)x, incx, (const double *)beta, y, incy);
// }

void getrfBatched(hipblasHandle_t handle, int n, float *Aarray[], int lda, int *PivotArray, int *infoArray, int batchSize) {
  hipblasSgetrfBatched(handle, n, Aarray, lda, PivotArray, infoArray, batchSize);
}

void getrfBatched(hipblasHandle_t handle, int n, double *Aarray[], int lda, int *PivotArray, int *infoArray, int batchSize) {
  hipblasDgetrfBatched(handle, n, Aarray, lda, PivotArray, infoArray, batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, float *Aarray[], int lda, int *PivotArray, float *Carray[], int ldc, int *infoArray, int batchSize) {
  hipblasSgetriBatched(handle, n, (const float **)Aarray, lda, PivotArray, Carray, ldc, infoArray, batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, double *Aarray[], int lda, int *PivotArray, double *Carray[], int ldc, int *infoArray, int batchSize) {
  hipblasDgetriBatched(handle, n, (const double **)Aarray, lda, PivotArray, Carray, ldc, infoArray, batchSize);
}


#endif
