#include "hip/hip_runtime.h"
#ifndef KERNELS_CU
#define KERNELS_CU

#include "kernels.h"

template<typename T>
__global__
void set_model_kernel(T *Y, T *y, T *mu, T *beta, T *alp, T *bet, int *nVars, int *lasso, int *step, int *done, int *act, int M, int N, int mod, int hact) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;
	if (ind == 0) {
		nVars[mod] = 0;
		lasso[mod] = 0;
		step[mod] = 0;
		done[mod] = 0;
		act[mod] = hact;
		alp[0] = 1;
		bet[0] = 0;
	}
	if (ind < M) {
		mu[mod * M + ind] = 0;
		y[mod * M + ind] = Y[ind * N + hact];
	}
	if (ind < N) {
		beta[mod * N + ind] = 0;
	}
}

template<typename T>
void set_model(T *Y, T *y, T *mu, T *beta, T *alp, T *bet, int *nVars, int *lasso, int *step, int *done, int *act, int M, int N, int mod, int hact, hipStream_t stream, dim3 blockDim) {
	dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
	set_model_kernel<T><<<gridDim, blockDim, 0, stream>>>(Y, y, mu, beta, alp, bet, nVars, lasso, step, done, act, M, N, mod, hact);
}

template void set_model<float>(float *Y, float *y, float *mu, float *beta, float *alp, float *bet, int *nVars, int *lasso, int *step, int *done, int *act, int M, int N, int mod, int hact, hipStream_t stream, dim3 blockDim);
template void set_model<double>(double *Y, double *y, double *mu, double *beta, double *alp, double *bet, int *nVars, int *lasso, int *step, int *done, int *act, int M, int N, int mod, int hact, hipStream_t stream, dim3 blockDim);

__global__
void check_kernel(int *nVars, int *step, int maxVariables, int maxSteps, int *done, int *ctrl, int numModels) {
	int mod = threadIdx.x + blockIdx.x * blockDim.x;
	if (mod < numModels) {
		if (nVars[mod] < maxVariables && step[mod] < maxSteps && !done[mod]) {
		}
		else {
			if (done[mod] == 0 || done[mod] == 1) {
				done[mod] = 2;
				ctrl[0] = 1;
			}
			else if (done[mod] == 2) {
				done[mod] = 3;
			}
		}
	}
}

void check(int *nVars, int *step, int maxVariables, int maxSteps, int *done, int *ctrl, int numModels) {
	int block = (numModels < 1024)? numModels: 1024;
	dim3 blockDim(block);
	dim3 gridDim((numModels + blockDim.x - 1) / blockDim.x);
	check_kernel<<<gridDim, blockDim>>>(nVars, step, maxVariables, maxSteps, done, ctrl, numModels);
}

template<typename T>
__global__
void mat_sub_kernel(T *a, T *b, T *c, int size) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;
	if (ind < size) {
		c[ind] = a[ind] - b[ind];
	}
}

template<typename T>
void mat_sub(T *a, T *b, T *c, int size, dim3 blockDim) {
	dim3 gridDim((size + blockDim.x - 1) / blockDim.x);
	mat_sub_kernel<T><<<gridDim, blockDim>>>(a, b, c, size);
}

template void mat_sub<float>(float *a, float *b, float *c, int size, dim3 blockDim);
template void mat_sub<double>(double *a, double *b, double *c, int size, dim3 blockDim);

template<typename T>
__global__
void exclude_kernel(T *absC, int *lVars, int *nVars, int *act, int M, int N, int numModels, T def) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;
	int mod = ind / M;
	ind -= mod * M;
	if (mod < numModels) {
		int ni = nVars[mod];
		if (ind == M - 1) {
			absC[mod * N + act[mod]] = def;
		}
		if (ind < ni) {
			int li = lVars[mod * M + ind];
			absC[mod * N + li] = def;
		}
	}
}

template<typename T>
void exclude(T *absC, int *lVars, int *nVars, int *act, int M, int N, int numModels, T def, dim3 blockDim) {
	dim3 gridDim((numModels * M + blockDim.x - 1) / blockDim.x);
	exclude_kernel<T><<<gridDim, blockDim>>>(absC, lVars, nVars, act, M, N, numModels, def);
}

template void exclude<float>(float *absC, int *lVars, int *nVars, int *act, int M, int N, int numModels, float def, dim3 blockDim);
template void exclude<double>(double *absC, int *lVars, int *nVars, int *act, int M, int N, int numModels, double def, dim3 blockDim);

__global__
void lasso_add_kernel(int *lasso, int *lVars, int *nVars, int *cidx, int M, int N, int numModels) {
	int mod = threadIdx.x + blockIdx.x * blockDim.x;
	if (mod < numModels) {
		if (!lasso[mod]) {
			int ni = nVars[mod];
			int id = cidx[mod];
			lVars[mod * M + ni] = id;
			nVars[mod] = ni + 1;
		}
	}
}

void lasso_add(int *lasso, int *lVars, int *nVars, int *cidx, int M, int N, int numModels, dim3 blockDim) {
	dim3 gridDim((numModels + blockDim.x - 1) / blockDim.x);
	lasso_add_kernel<<<gridDim, blockDim>>>(lasso, lVars, nVars, cidx, M, N, numModels);
}

template<typename T>
__global__
void gather_add_kernel(T *XA, T *XA1, T *X, int *lVars, int ni, int M, int N, int mod) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;
	if (ind < M) {
		XA[(ni - 1) * M + ind] = XA1[(ni - 1) * M + ind] = X[ind * N + lVars[mod * M + ni - 1]];
	}
}

template<typename T>
__global__
void gather_del_kernel(T *XA, T *XA1, T *X, int ni, int drop, int M, int N, int mod) {
	int mj = threadIdx.x + blockIdx.x * blockDim.x;
	int mi = mj / M;
	mj -= mi * M;
	mi += drop;
	if (mi >= drop && mi < ni && mj < M) {
		XA[mi * M + mj] = XA1[(mi + 1) * M + mj];
	}
}

template<typename T>
__global__
void gather_cop_kernel(T *XA, T *XA1, T *X, int ni, int drop, int M, int N, int mod) {
	int mj = threadIdx.x + blockIdx.x * blockDim.x;
	int mi = mj / M;
	mj -= mi * M;
	mi += drop;
	if (mi >= drop && mi < ni && mj < M) {
		XA1[mi * M + mj] = XA[mi * M + mj];
	}
}

template<typename T>
void gather(T *XA, T *XA1, T *X, int *lVars, int ni, int lassoCond, int drop, int M, int N, int mod, hipStream_t stream) {
	if (!lassoCond) {
		gather_add_kernel<T><<<1, M, 0, stream>>>(XA, XA1, X, lVars, ni, M, N, mod);
	}
	else {
		dim3 blockDim(1024);
		dim3 gridDim(((ni - drop) * M + blockDim.x - 1) / blockDim.x);
		gather_del_kernel<T><<<gridDim, blockDim, 0, stream>>>(XA, XA1, X, ni, drop, M, N, mod);
		gather_cop_kernel<T><<<gridDim, blockDim, 0, stream>>>(XA, XA1, X, ni, drop, M, N, mod);
	}
}

template void gather<float>(float *XA, float *XA1, float *X, int *lVars, int ni, int lassoCond, int drop, int M, int N, int mod, hipStream_t stream);
template void gather<double>(double *XA, double *XA1, double *X, int *lVars, int ni, int lassoCond, int drop, int M, int N, int mod, hipStream_t stream);

template<typename T>
__global__
void gammat_kernel(T *gamma_tilde, T *beta, T *betaOls, int *lVars, int *nVars, int *lasso, int M, int N, int numModels) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;
	int mod = ind / M;
	ind -= mod * M;
	if (ind < M && mod < numModels) {
		int ni = nVars[mod];
		if (ind < ni - 1) {
			int si = lVars[mod * M + ind];
			T val = beta[mod * N + si] / (beta[mod * N + si] - betaOls[mod * M + ind]);
			val = (val <= 0)? inf: val;
			gamma_tilde[mod * M + ind] = val;
		}
		else if (ind == 0 && ni - 1 <= 0) {
			gamma_tilde[mod * M + ind] = inf;
		}
		if (ind == 0 && lasso[mod]) {
			lasso[mod] = 0;
		}
	}
}

template<typename T>
void gammat(T *gamma_tilde, T *beta, T *betaOls, int *lVars, int *nVars, int *lasso, int M, int N, int numModels, dim3 blockDim) {
	dim3 gridDim((numModels * M + blockDim.x - 1) / blockDim.x);
	gammat_kernel<T><<<gridDim, blockDim>>>(gamma_tilde, beta, betaOls, lVars, nVars, lasso, M, N, numModels);
}

template void gammat<float>(float *gamma_tilde, float *beta, float *betaOls, int *lVars, int *nVars, int *lasso, int M, int N, int numModels, dim3 blockDim);
template void gammat<double>(double *gamma_tilde, double *beta, double *betaOls, int *lVars, int *nVars, int *lasso, int M, int N, int numModels, dim3 blockDim);

template<typename T>
__global__
void set_gamma_kernel(T *gamma, T *gamma_tilde, T *r, int *dropidx, int *lasso, int *nVars, int maxVariables, int M, int numModels) {
	int mod = threadIdx.x + blockIdx.x * blockDim.x;
	if (mod < numModels) {
		T gamma_t = gamma_tilde[mod * M + dropidx[mod] - 1];
		T gamma_val = r[mod];
		if (nVars[mod] == maxVariables) {
			gamma[mod] = 1;
		}
		else if (gamma_t < gamma_val) {
			lasso[mod] = 1;
			gamma[mod] = gamma_t;
		}
		else {
			gamma[mod] = gamma_val;
		}
	}
}

template<typename T>
void set_gamma(T *gamma, T *gamma_tilde, T *r, int *dropidx, int *lasso, int *nVars, int maxVariables, int M, int numModels, dim3 blockDim) {
	dim3 gridDim((numModels + blockDim.x - 1) / blockDim.x);
	set_gamma_kernel<T><<<gridDim, blockDim>>>(gamma, gamma_tilde, r, dropidx, lasso, nVars, maxVariables, M, numModels);
}

template void set_gamma<float>(float *gamma, float *gamma_tilde, float *r, int *dropidx, int *lasso, int *nVars, int maxVariables, int M, int numModels, dim3 blockDim);
template void set_gamma<double>(double *gamma, double *gamma_tilde, double *r, int *dropidx, int *lasso, int *nVars, int maxVariables, int M, int numModels, dim3 blockDim);

template<typename T>
__global__
void update_kernel(T *beta, T *mu, T *d, T *betaOls, T *gamma, int *lVars, int *nVars, int M, int N, int numModels) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;
	int mod = ind / M;
	ind -= mod * M;
	if (mod < numModels) {
		int ni = nVars[mod];
		T gamma_val = gamma[mod];
		if (ind < M) {
			mu[mod * M + ind] += gamma_val * d[mod * M + ind];
			if (ind < ni) {
				int si = lVars[mod * M + ind];
				beta[mod * N + si] += gamma_val * (betaOls[mod * M + ind] - beta[mod * N + si]);
			}
		}
	}
}

template<typename T>
void update(T *beta, T *mu, T *d, T *betaOls, T *gamma, int *lVars, int *nVars, int M, int N, int numModels, dim3 blockDim) {
	dim3 gridDim((numModels * M + blockDim.x - 1) / blockDim.x);
	update_kernel<T><<<gridDim, blockDim>>>(beta, mu, d, betaOls, gamma, lVars, nVars, M, N, numModels);
}

template void update<float>(float *beta, float *mu, float *d, float *betaOls, float *gamma, int *lVars, int *nVars, int M, int N, int numModels, dim3 blockDim);
template void update<double>(double *beta, double *mu, double *d, double *betaOls, double *gamma, int *lVars, int *nVars, int M, int N, int numModels, dim3 blockDim);

__global__
void drop_kernel(int *lVars, int *dropidx, int *nVars, int *lasso, int M, int numModels) {
	int mod = blockIdx.x;
	int ind = threadIdx.x;
	if (mod < numModels && lasso[mod]) {
		int ni = nVars[mod];
		int drop = dropidx[mod] - 1;
		if (ind < ni && ind > drop) {
			int val = lVars[mod * M + ind];
			__syncthreads();
			lVars[mod * M + ind - 1] = val;
		}
		else return;
		if (ind == ni - 1) nVars[mod] = ni - 1;
	}
	else return;
}

void drop(int *lVars, int *dropidx, int *nVars, int *lasso, int M, int numModels) {
		drop_kernel<<<numModels, M>>>(lVars, dropidx, nVars, lasso, M, numModels);
}

template<typename T>
__global__
void final_kernel(T *a1, T *a2, T *cmax, T *r, int *step, int *done, int numModels, T g) {
	int mod = threadIdx.x + blockIdx.x * blockDim.x;
	if (mod < numModels) {
		step[mod] += 1;
		T a1_val = cmax[mod], a2_val = sqrt(r[mod]);
		if (step[mod] > 1) {
			T G = -(a2_val - a2[mod]) / (a1_val - a1[mod]);
			if (G < g) {
				if (!done[mod]) done[mod] = 1;
				return;
			}
		}
		a1[mod] = a1_val;
		a2[mod] = a2_val;
	}
}

template<typename T>
void final(T *a1, T *a2, T *cmax, T *r, int *step, int *done, int numModels, T g, dim3 blockDim) {
		dim3 gridDim((numModels + blockDim.x - 1) / blockDim.x);
		final_kernel<<<gridDim, blockDim>>>(a1, a2, cmax, r, step, done, numModels, g);
}

template void final<float>(float *a1, float *a2, float *cmax, float *r, int *step, int *done, int numModels, float g, dim3 blockDim);
template void final<double>(double *a1, double *a2, double *cmax, double *r, int *step, int *done, int numModels, double g, dim3 blockDim);

#endif
