#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "utilities.h"
#include "blas.h"
#include "kernels.h"

typedef float precision;

double flopCounter(int M, int N, int numModels, int *hNVars) {
	double flop = 0;
	// r = y - mu
	flop += (double) M * (double) numModels;
	// c = X' * r
	flop += 2.0 * (double) M * (double) N * (double) numModels;
	// abs(c)
	flop += (double) N * (double) numModels;
	for (int i = 0; i < numModels; i++) {
		// G = X(:, A)' * X(:, A)
		flop += 2.0 * (double) hNVars[i] * (double) M * (double) hNVars[i];
		// b_OLS = G\(X(:,A)'*y)
		flop += 2.0 * (double) M * (double) hNVars[i] + 2.0 * (double) hNVars[i] * (double) hNVars[i];
		// Inverse ops for G
		flop += (2.0 / 3.0) * (double) hNVars[i] * (double) hNVars[i] * (double) hNVars[i];
		// d = X(: , A) * b_OLS - mu
		flop += 2.0 * (double) M * (double) hNVars[i] + (double) M;
		// gamma_tilde
		flop += 2.0 * (double) hNVars[i];
		// b update
		flop += 3.0 * (double) hNVars[i];
		// norm1
		flop += 2.0 * (double) hNVars[i];
	}
	// cd = X'*d
	flop += 2.0 * (double) M * (double) N * (double) numModels;
	// gamma
	flop += 6.0 * (double) N * (double) numModels;
	// mu update
	flop += 2.0 * (double) M * (double) numModels;
	// norm2
	flop += 3.0 * (double) M * (double) numModels;
	// norm2 sqrt, step, sb, err
	flop += 4.0 * (double) numModels;
	// G
	flop += 3.0 * (double) M * (double) numModels;
	return flop;
}

int main(int argc, char *argv[]) {
	if (argc < 9) {
		printf("Insufficient parameters, required 8! (flatMriPath, numModels, numStreams, max l1, min l2, min g, max vars, max steps)\nInput 0 for a parameter to use it's default value!\n");
		return 0;
	}

	// Reading flattened mri image
	precision *X, *Y;
	IntegerTuple tuple = read_flat_mri<precision>(argv[1], X, Y);
	int M = tuple.M, N = tuple.N;
	printf("Read FMRI Data X of shape: (%d,%d)\n", M, N);
	printf("Read FMRI Data Y of shape: (%d,%d)\n", M, N);

	// Number of models to solve in ||l
	int numModels = atoi(argv[2]);
	numModels = (numModels == 0)? 512: numModels;
	int totalModels = N;
	printf("Total number of models: %d\n", totalModels);
	printf("Number of models in ||l: %d\n", numModels);

	int numStreams = atoi(argv[3]);
	numStreams = (numStreams == 0)? 8: numStreams;
	numStreams = pow(2, int(log(numStreams) / log(2)));
	printf("Number of streams: %d\n", numStreams);

	precision l1 = atof(argv[4]);
	l1 = (l1 == 0)? 1000: l1;
	printf("Max L1: %f\n", l1);

	precision l2 = atof(argv[5]);
	printf("Min L2: %f\n", l2);

	precision g = atof(argv[6]);
	g = (g == 0)? 0.1: g;
	printf("Lambda: %f\n", g);

	int maxVariables = atoi(argv[7]);
	maxVariables = (maxVariables == 0)? N - 1: maxVariables;
	maxVariables = min(min(M, N - 1), maxVariables);
	printf("Max Variables: %d\n", maxVariables);

	int maxSteps = atoi(argv[8]);
	maxSteps = (maxSteps == 0)? 8 * maxVariables: maxSteps;
	maxSteps = min(8 * maxVariables, maxSteps);
	printf("Max Steps: %d\n", maxSteps);

	// Computimal optimal block sizes
	int bN = optimalBlock1D(N);
	int bM = optimalBlock1D(M);
	int bModM = optimalBlock1D(numModels * M);
	int bMM = optimalBlock1D(M * M);
	int bModN = optimalBlock1D(numModels * N);
	int bMod = optimalBlock1D(numModels);
		
	// Declare all lars variables
	int *nVars, *step, *lasso, *done, *cidx, *act, *dropidx;
	int *pivot, *info, *intBuf;
	int *lVars;
	int *hNVars, *hStep, *hdone, *hact, *hLasso, *hDropidx;
	precision *cmax, *a1, *a2, *lambda, *gamma;
	precision *y, *mu, *r, *betaOls, *d, *gamma_tilde, *buf;
	precision *beta, *c, *cd, *beta_prev;
	precision alp = 1, bet = 0;
	precision *XA[numModels], *XA1[numModels], *G[numModels], *I[numModels], **dXA, **dG, **dI;
	precision *ha1, *ha2, *hlambda;
	double corr_alp = 1, corr_bet = 0;
	double *corr_beta, *corr_sb, *corr_y, *corr_tmp, *corr_betaols, *corr_yh, *corr_z;
	double *corr_XA[numModels], *corr_G[numModels], *corr_I[numModels], **corr_dXA, **corr_dG, **corr_dI;

	// Initialize all lars variables
	init_var<int>(nVars, numModels);
	init_var<int>(step, numModels);
	init_var<int>(lasso, numModels);
	init_var<int>(done, numModels);
	init_var<int>(cidx, numModels);
	init_var<int>(act, numModels);
	init_var<int>(dropidx, numModels);
	
	init_var<int>(pivot, M * numModels * M);
	init_var<int>(info, M * numModels);
	init_var<int>(intBuf, numModels * 128);
	
	init_var<int>(lVars, numModels * M);
	
	hNVars = new int[numModels];
	hStep = new int[numModels];
	hdone = new int[numModels];
	hact = new int[numModels];
	hLasso = new int[numModels];
	hDropidx = new int[numModels];
	
	init_var<precision>(cmax, numModels);
	init_var<precision>(a1, numModels);
	init_var<precision>(a2, numModels);
	init_var<precision>(lambda, numModels);
	init_var<precision>(gamma, numModels);

	init_var<precision>(y, numModels * M);
	init_var<precision>(mu, numModels * M);
	init_var<precision>(r, numModels * M);
	init_var<precision>(betaOls, numModels * M);
	init_var<precision>(d, numModels * M);
	init_var<precision>(gamma_tilde, numModels);
	init_var<precision>(buf, numModels * 128);
		
	init_var<precision>(beta, numModels * N);
	init_var<precision>(c, numModels * N);
	init_var<precision>(cd, numModels * N);
	init_var<precision>(beta_prev, numModels * N);

	init_var<double>(corr_beta, numModels * M);
	init_var<double>(corr_sb, numModels * M);
	init_var<double>(corr_y, numModels * M);
	init_var<double>(corr_tmp, numModels * M);
	init_var<double>(corr_betaols, numModels * M);
	init_var<double>(corr_yh, numModels * M);
	init_var<double>(corr_z, numModels * M);

	ha1 = new precision[numModels];
	ha2 = new precision[numModels];
	hlambda = new precision[numModels];

	for (int i = 0; i < numModels; i++) {
		init_var<precision>(XA[i], M * M);
		init_var<precision>(XA1[i], M * M);
		init_var<precision>(G[i], M * M);
		init_var<precision>(I[i], M * M);

		init_var<double>(corr_XA[i], M * M);
		init_var<double>(corr_G[i], M * M);
		init_var<double>(corr_I[i], M * M);
	}
	
	hipMalloc(&dXA, numModels * sizeof(precision *));
	hipMemcpy(dXA, XA, numModels * sizeof(precision *), hipMemcpyHostToDevice);	
	hipMalloc(&dG, numModels * sizeof(precision *));
	hipMemcpy(dG, G, numModels * sizeof(precision *), hipMemcpyHostToDevice);
	hipMalloc(&dI, numModels * sizeof(precision *));
	hipMemcpy(dI, I, numModels * sizeof(precision *), hipMemcpyHostToDevice);

	hipMalloc(&corr_dXA, numModels * sizeof(double *));
	hipMemcpy(corr_dXA, corr_XA, numModels * sizeof(double *), hipMemcpyHostToDevice);
	hipMalloc(&corr_dG, numModels * sizeof(double *));
	hipMemcpy(corr_dG, corr_G, numModels * sizeof(double *), hipMemcpyHostToDevice);
	hipMalloc(&corr_dI, numModels * sizeof(double *));
	hipMemcpy(corr_dI, corr_I, numModels * sizeof(double *), hipMemcpyHostToDevice);

	precision **batchXA[maxVariables], **batchG[maxVariables], **batchI[maxVariables], **dBatchXA[maxVariables], **dBatchG[maxVariables], **dBatchI[maxVariables];
	for (int i = 0; i < maxVariables; i++) {
		batchXA[i] = new precision *[numModels];
		batchG[i] = new precision *[numModels];
		batchI[i] = new precision *[numModels];
		hipMalloc(&dBatchXA[i], numModels * sizeof(precision *));
		hipMalloc(&dBatchG[i], numModels * sizeof(precision *));
		hipMalloc(&dBatchI[i], numModels * sizeof(precision *));
	}
	int batchLen[maxVariables];

	hipblasHandle_t hnd;
	hipblasCreate(&hnd);
	hipStream_t streams[numStreams];
	for (int i = 0; i < numStreams; i++) hipStreamCreate(&streams[i]);

	for (int i = 0; i < numModels; i++) set_model<precision>(Y, y, mu, beta, a1, a2, lambda, nVars, lasso, step, done, act, M, N, i, i, streams[i & (numStreams - 1)], *(new dim3(bN)));
	hipDeviceSynchronize();

	GpuTimer timer;
	std::ofstream stepf("step.csv"), nvarsf("nvars.csv"), a1f("a1.csv"), a2f("a2.csv"), lambdaf("G.csv"), betaf("beta.csv");

	int top = numModels;
	double totalFlop = 0;
	double times[25] = {0};
	int e = 0;
	int completed_count = 0;
	std::map<int, int> completed;
	while (true) {
		int t = 0;

		timer.start();
		check(nVars, step, a1, a2, lambda, maxVariables, maxSteps, l1, l2, g, done, numModels);
		int ctrl = 0;
		hipMemcpy(hdone, done, numModels * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(hact, act, numModels * sizeof(int), hipMemcpyDeviceToHost);
		for (int i = 0; i < numModels; i++) {
			if (hdone[i] && !completed[hact[i]]) {
				ctrl = 1;
				break;
			}
		}

		if (ctrl) {
			hipMemcpy(hStep, step, numModels * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(hNVars, nVars, numModels * sizeof(int), hipMemcpyDeviceToHost);

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					copyUp<precision>(corr_XA[i], XA[i], hNVars[i] * M, streams[s & (numStreams - 1)]);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					copyUp<precision>(corr_y + i * M, y + i * M, M, streams[s & (numStreams - 1)]);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					computeSign<precision>(corr_sb + i * M, beta + i * N, beta_prev + i * N, lVars + i * M, hNVars[i], streams[s & (numStreams - 1)]);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					gemm(hnd, HIPBLAS_OP_T, HIPBLAS_OP_N, hNVars[i], hNVars[i], M, &corr_alp, corr_XA[i], M, corr_XA[i], M, &corr_bet, corr_G[i], hNVars[i]);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					getrfBatched(hnd, hNVars[i], corr_dG + i, hNVars[i], pivot + i * M, info + i, 1);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					getriBatched(hnd, hNVars[i], corr_dG + i, hNVars[i], pivot + i * M, corr_dI + i, hNVars[i], info + i, 1);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					gemv(hnd, HIPBLAS_OP_T, M, hNVars[i], &corr_alp, corr_XA[i], M, corr_y + i * M, 1, &corr_bet, corr_tmp + i * M, 1);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					gemv(hnd, HIPBLAS_OP_N, hNVars[i], hNVars[i], &corr_alp, corr_I[i], hNVars[i], corr_tmp + i * M, 1, &corr_bet, corr_betaols + i * M, 1);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					gemv(hnd, HIPBLAS_OP_N, M, hNVars[i], &corr_alp, corr_XA[i], M, corr_betaols + i * M, 1, &corr_bet, corr_yh + i * M, 1);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					gemv(hnd, HIPBLAS_OP_N, hNVars[i], hNVars[i], &corr_alp, corr_I[i], hNVars[i], corr_sb + i * M, 1, &corr_bet, corr_tmp + i * M, 1);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
					gemv(hnd, HIPBLAS_OP_N, M, hNVars[i], &corr_alp, corr_XA[i], M, corr_tmp + i * M, 1, &corr_bet, corr_z + i * M, 1);
					s++;
				}
			}
			hipDeviceSynchronize();

			for (int i = 0, s = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					correct<precision>(corr_beta + i * M, corr_betaols + i * M, corr_tmp + i * M, corr_y + i * M, corr_yh + i * M, corr_z + i * M, a1 + i, a2 + i, lambda + i, l2, g, hNVars[i], M, streams[s & (numStreams - 1)]);
					s++;
				}
			}
			hipDeviceSynchronize();

			hipMemcpy(ha1, a1, numModels * sizeof(precision), hipMemcpyDeviceToHost);
			hipMemcpy(ha2, a2, numModels * sizeof(precision), hipMemcpyDeviceToHost);
			hipMemcpy(hlambda, lambda, numModels * sizeof(precision), hipMemcpyDeviceToHost);

			for (int i = 0; i < numModels; i++) {
				if (hdone[i] && !completed[hact[i]]) {
					completed[hact[i]] = 1;
					completed_count++;
					stepf << hact[i] << ", " << hStep[i] << "\n";
					nvarsf << hact[i] << ", " << hNVars[i] << "\n";
					a1f << hact[i] << ", " << ha1[i] << "\n";
					a2f << hact[i] << ", " << ha2[i] << "\n";
					lambdaf << hact[i] << ", " << hlambda[i] << "\n";
					int hlVars[hNVars[i]];
					double hbeta[hNVars[i]];
					hipMemcpy(hlVars, lVars + i * M, hNVars[i] * sizeof(int), hipMemcpyDeviceToHost);
					hipMemcpy(hbeta, corr_beta + i * M, hNVars[i] * sizeof(double), hipMemcpyDeviceToHost);
					for (int j = 0; j < hNVars[i]; j++) betaf << hact[i] << ", " << hlVars[j] << ", " << hbeta[j] << "\n";
				}
			}

			for (int i = 0, s = 0; i < numModels && top < totalModels; i++) {
				if (hdone[i] && completed[hact[i]]) {
					set_model<precision>(Y, y, mu, beta, a1, a2, lambda, nVars, lasso, step, done, act, M, N, i, top++, streams[s & (numStreams - 1)], *(new dim3(bN)));
					s++;
					hdone[i] = 0;
				}
			}
			hipDeviceSynchronize();
		}
		printf("\rCompleted %d models", completed_count);
		if (completed_count == totalModels) {
			break;
		}
		timer.stop();
		times[t++] += timer.elapsed();

		timer.start();
		drop(lVars, dropidx, nVars, lasso, M, numModels);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();

		timer.start();
		mat_sub<precision>(y, mu, r, numModels * M, *(new dim3(bModM)));
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		hipblasSetStream(hnd, NULL);
		gemm(hnd, HIPBLAS_OP_N, HIPBLAS_OP_N, N, numModels, M, &alp, X, N, r, M, &bet, c, N);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		exclude<precision>(c, lVars, nVars, act, M, N, numModels, 0, *(new dim3(bModM)));
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		fabsMaxReduce<precision>(c, cmax, buf, cidx, intBuf, numModels, N);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();

		timer.start();
		lasso_add(lasso, lVars, nVars, cidx, M, N, numModels, *(new dim3(bMod)));
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();

		timer.start();
		hipMemcpy(hNVars, nVars, numModels * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(hLasso, lasso, numModels * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(hDropidx, dropidx, numModels * sizeof(int), hipMemcpyDeviceToHost);
		int maxVar = hNVars[0];
		for (int i = 0; i < maxVariables; i++) batchLen[i] = 0;
		for (int i = 0; i < numModels; i++) {
			if (hNVars[i] > maxVar) maxVar = hNVars[i];
			batchXA[hNVars[i]][batchLen[hNVars[i]]] = XA[i];
			batchG[hNVars[i]][batchLen[hNVars[i]]] = G[i];
			batchI[hNVars[i]][batchLen[hNVars[i]]] = I[i];
			batchLen[hNVars[i]]++;
		}
		for (int i = 0; i < maxVariables; i++) {
			if (batchLen[i] > 0) {
				hipMemcpy(dBatchXA[i], batchXA[i], batchLen[i] * sizeof(precision *), hipMemcpyHostToDevice);
				hipMemcpy(dBatchG[i], batchG[i], batchLen[i] * sizeof(precision *), hipMemcpyHostToDevice);
				hipMemcpy(dBatchI[i], batchI[i], batchLen[i] * sizeof(precision *), hipMemcpyHostToDevice);
			}
		}
		hipDeviceSynchronize();
		timer.stop();
		times[0] += timer.elapsed();

		timer.start();
		for (int i = 0, s = 0; i < numModels; i++) {
			gather<precision>(XA[i], XA1[i], X, lVars, hNVars[i], hLasso[i], hDropidx[i], M, N, i, streams[s & (numStreams - 1)]);
			s++;
		}
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();

		timer.start();
		for (int i = 0, s = 0; i < maxVariables; i++) {
			if (batchLen[i] > 0) {
				hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
				gemmBatched(hnd, HIPBLAS_OP_T, HIPBLAS_OP_N, i, i, M, &alp, dBatchXA[i], M, dBatchXA[i], M, &bet, dBatchG[i], i, batchLen[i]);
				s++;
			}
		}
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		XAyBatched<precision>(dXA, y, r, nVars, M, numModels);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		for (int i = 0, s = 0; i < maxVariables; i++) {
			if (batchLen[i] > 0) {
				hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
				getrfBatched(hnd, i, dBatchG[i], i, pivot + i * numModels * M, info + i * numModels, batchLen[i]);
				s++;
			}
		}
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		for (int i = 0, s = 0; i < maxVariables; i++) {
			if (batchLen[i] > 0) {
				hipblasSetStream(hnd, streams[s & (numStreams - 1)]);
				getriBatched(hnd, i, dBatchG[i], i, pivot + i * numModels * M, dBatchI[i], i, info + i * numModels, batchLen[i]);
				s++;
			}
		}
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		IrBatched<precision>(dI, r, betaOls, nVars, M, numModels, maxVar);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		XAbetaOlsBatched<precision>(dXA, betaOls, d, nVars, M, numModels, maxVar);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();

		timer.start();
		mat_sub<precision>(d, mu, d, numModels * M, *(new dim3(bModM)));
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		gammat<precision>(gamma_tilde, beta, betaOls, dropidx, lVars, nVars, lasso, M, N, numModels, *(new dim3(bModM)));
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		hipblasSetStream(hnd, NULL);
		gemm(hnd, HIPBLAS_OP_N, HIPBLAS_OP_N, N, numModels, M, &alp, X, N, d, M, &bet, cd, N);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		cdMinReduce<precision>(c, cd, cmax, r, buf, numModels, N);
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		set_gamma<precision>(gamma, gamma_tilde, r, lasso, nVars, maxVariables, M, numModels, *(new dim3(bMod)));
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		timer.start();
		update<precision>(beta, beta_prev, mu, d, betaOls, gamma, dXA, y, a1, a2, lambda, lVars, nVars, step, M, N, numModels, l1, *(new dim3(bMod)));
		hipDeviceSynchronize();
		timer.stop();
		times[t++] += timer.elapsed();
		
		totalFlop += flopCounter(M, N, numModels, hNVars);
		e++;
	}

	stepf.close();
	nvarsf.close();
	a1f.close();
	a2f.close();
	lambdaf.close();
	betaf.close();

	// Statistics
	double transferTime = times[0];
	double execTime = 0;
	for (int i = 1; i < 25; i++) execTime += times[i];
	printf("\n");

 	for (int i = 0; i < 25; i++) {
 		printf("Kernel %2d time = %10.4f\n", i, times[i]);
 	}
 	printf("Execution time(s) = %f\n", execTime * 1.0e-3);
 	printf("Transfer time(s) = %f\n", transferTime * 1.0e-3);
	printf("Total Flop count(gflop) = %f\n", totalFlop * 1.0e-9);
	printf("Execution Flops(gflops) = %f\n", (totalFlop * 1.0e-9) / (execTime * 1.0e-3));

	hipFree(nVars);
	hipFree(step);
	hipFree(lasso);
	hipFree(done);
	hipFree(cidx);
	hipFree(act);
	hipFree(dropidx);
	
	hipFree(pivot);
	hipFree(info);
	hipFree(intBuf);
	
	hipFree(lVars);
	
	hipFree(cmax);
	hipFree(a1);
	hipFree(a2);
	hipFree(lambda);
	hipFree(gamma);

	hipFree(y);
	hipFree(mu);
	hipFree(r);
	hipFree(betaOls);
	hipFree(d);
	hipFree(gamma_tilde);
	hipFree(buf);
	
	hipFree(beta);
	hipFree(c);
	hipFree(cd);
	hipFree(beta_prev);

	hipFree(corr_beta);
	hipFree(corr_sb);
	hipFree(corr_y);
	hipFree(corr_tmp);
	hipFree(corr_betaols);
	hipFree(corr_yh);
	hipFree(corr_z);

	for (int i = 0; i < numModels; i++) {
		hipFree(XA[i]);
		hipFree(XA1[i]);
		hipFree(G[i]);
		hipFree(I[i]);

		hipFree(corr_XA[i]);
		hipFree(corr_G[i]);
		hipFree(corr_I[i]);
	}

	for (int i = 0; i < maxVariables; i++) {
		hipFree(dBatchXA[i]);
		hipFree(dBatchG[i]);
		hipFree(dBatchI[i]);
	}

	for (int i = 0; i < numStreams; i++) hipStreamDestroy(streams[i]);
		
	hipFree(dXA);
	hipFree(dG);
	hipFree(dI);

	hipFree(corr_dXA);
	hipFree(corr_dG);
	hipFree(corr_dI);

	hipblasDestroy(hnd);

	return 0;
}
