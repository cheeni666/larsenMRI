#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "utilities.h"
#include "blas.h"
#include "kernels.h"

typedef float precision;

template<typename T>
void printDeviceVar(T *var, int size, int *ind, int numInd) {
    T *hVar = new T[size];
    hipMemcpy(hVar, var, size * sizeof(T), hipMemcpyDeviceToHost);
    for (int i = 0; i < numInd; i++) {
        printf("%f:%d\n", hVar[ind[i]], hVar[ind[i]]);
    }
}

int optimalBlock1D(int problemSize) {
    int blockSize, minR = inf;
    for (int i = 1024; i >= 256; i -= 32) {
        int ans = problemSize % i;
        if (ans < minR) {
            minR = ans;
            blockSize = i;
        }
    }
    return blockSize;
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        printf("Insufficient parameters, required 3! (flatMriPath, numModels, numStreams)\n");
        return 0;
    }

    // Reading flattened mri image.
    precision *X, *Y;
    IntegerTuple tuple = read_flat_mri<precision>(argv[1], X, Y);
    int M = tuple.M, N = tuple.N;
    printf("Read FMRI Data X of shape: (%d,%d)\n", M, N);
    printf("Read FMRI Data Y of shape: (%d,%d)\n", M, N);

    // Number of models to solve in ||l.
    int numModels = str_to_int(argv[2]);
    int totalModels = N;
    printf("Total number of models: %d\n", totalModels);
    printf("Number of models in ||l: %d\n", numModels);

    int numStreams = str_to_int(argv[3]);
    numStreams = pow(2, int(log(numStreams) / log(2)));
    printf("Number of streams: %d\n", numStreams);
    
    // Declare all lars variables.
    int *nVars, *step, *lasso, *done, *lVars, *cidx, *act, *dropidx;
    int *pivot, *info, *ctrl, *hNVars, *hctrl, *hdone, *hact;
    precision *alp[numModels], *bet[numModels], *ha1, *ha2;
    precision *y, *mu, *r, *beta, *c, *absC, *cmax, *betaOls, *d, *gamma, *cd, *a1, *a2;
    precision *XA[numModels], *G[numModels], *I[numModels], **dG, **dI;

    // Initialize all lars variables.
    init_var<int>(nVars, numModels);
    init_var<int>(step, numModels);
    init_var<int>(lasso, numModels);
    init_var<int>(done, numModels);
    init_var<int>(lVars, numModels * M);
    init_var<int>(cidx, numModels);
    init_var<int>(dropidx, numModels);
    init_var<int>(act, numModels);

    init_var<int>(pivot, numModels * M);
    init_var<int>(info, numModels);
    init_var<int>(ctrl, 2);

    init_var<precision>(y, numModels * M);
    init_var<precision>(mu, numModels * M);
    init_var<precision>(r, numModels * M);
    init_var<precision>(beta, numModels * N);
    init_var<precision>(c, numModels * N);
    init_var<precision>(absC, numModels * N);
    init_var<precision>(cmax, numModels);
    init_var<precision>(betaOls, numModels * M);
    init_var<precision>(d, numModels * M);
    init_var<precision>(gamma, numModels * M);
    init_var<precision>(cd, numModels * N);
    init_var<precision>(a1, numModels);
    init_var<precision>(a2, numModels);

    int maxVariables = min(M, N - 1);
    int maxSteps = 8 * maxVariables;

    int top = numModels;
    double totalFlop = 0;
    double totalTime = 0;

    int bN = optimalBlock1D(N);
    int bM = optimalBlock1D(M);
    int bModM = optimalBlock1D(numModels * M);
    int bModN = optimalBlock1D(numModels * N);
    int bMod = optimalBlock1D(numModels);

    hipblasHandle_t hnd;
    hipblasCreate(&hnd);
    hipStream_t streams[numStreams];
    hipblasSetPointerMode(hnd, HIPBLAS_POINTER_MODE_DEVICE);

    hNVars = new int[numModels];
    hctrl = new int[2];
    hact = new int[numModels];
    hdone = new int[numModels];
    ha1 = new precision[numModels];
    ha2 = new precision[numModels];
    for (int i = 0; i < numModels; i++) {
        init_var<precision>(XA[i], M * M);
        init_var<precision>(G[i], M * M);
        init_var<precision>(I[i], M * M);
        init_var<precision>(alp[i], 1);
        init_var<precision>(bet[i], 1);
    }
    hipMalloc(&dG, numModels * sizeof(precision *));
    hipMemcpy(dG, G, numModels * sizeof(precision *), hipMemcpyHostToDevice);
    hipMalloc(&dI, numModels * sizeof(precision *));
    hipMemcpy(dI, I, numModels * sizeof(precision *), hipMemcpyHostToDevice);

    for (int i = 0; i < numStreams; i++)
        hipStreamCreate(&streams[i]);

    for (int i = 0; i < numModels; i++) {
        set_model<precision>(Y, y, mu,
                             beta, alp[i], bet[i],
                             nVars, lasso, step,
                             done, act, M, N,
                             i, i, streams[i & (numStreams - 1)],
                             *(new dim3(bN)));
    }
    hipDeviceSynchronize();

    Debug<precision> debug[totalModels];
    GpuTimer timer;

    printf("\rStack top at %d", top);
    while (true) {
        hipMemset(ctrl, 0, 2 * sizeof(int));
        check(nVars, step, maxVariables,
              maxSteps, done, ctrl,
              numModels);
        hipMemcpy(hctrl, ctrl, 2 * sizeof(int), hipMemcpyDeviceToHost);
        if (hctrl[1] == 1) {
            hipMemcpy(ha1, a1, numModels * sizeof(precision), hipMemcpyDeviceToHost);
            hipMemcpy(ha2, a2, numModels * sizeof(precision), hipMemcpyDeviceToHost);
            hipMemcpy(hdone, done, numModels * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hNVars, nVars, numModels * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hact, act, numModels * sizeof(int), hipMemcpyDeviceToHost);
            for (int i = 0; i < numModels; i++) {
                if (hdone[i]) {
                    if (top < totalModels) {
                        set_model<precision>(Y, y, mu,
                                             beta, alp[i], bet[i],
                                             nVars, lasso, step,
                                             done, act, M, N,
                                             i, top++, streams[i & (numStreams - 1)],
                                             *(new dim3(bN)));
                        printf("\rStack top at %d", top);
                    }
                    if (debug[hact[i]].nVars == -1) {
                        debug[hact[i]].nVars = hNVars[i];
                        debug[hact[i]].a1 = ha1[i];
                        debug[hact[i]].a2 = ha2[i];
                    }
                }
            }
        }
        if (hctrl[0] == 0) {
            break;
        }
        timer.start();
        mat_sub<precision>(y, mu, r,
                           numModels * M, *(new dim3(bModM)));
        hipDeviceSynchronize();
        hipblasSetStream(hnd, NULL);
        gemm(hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 N, numModels, M,
                 alp[0], X, N,
                 r, M, bet[0],
                 c, N);
        hipDeviceSynchronize();
        exclude<precision>(c, cmax, lVars, nVars,
                           act, M, N,
                           numModels, 0, *(new dim3(bModM)));
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            amaxFabs(c + i * N, cmax + i, N, streams[i & (numStreams - 1)], *(new dim3(1024)));
        }
        hipDeviceSynchronize();
        set_cidx<precision>(cmax, cidx, c,
                            N, numModels, *(new dim3(bModN)));
        hipDeviceSynchronize();
        lasso_add(lasso, lVars, nVars,
                  cidx, M, N,
                  numModels, *(new dim3(bMod)));
        hipDeviceSynchronize();
        hipMemcpy(hNVars, nVars, numModels * sizeof(int), hipMemcpyDeviceToHost);
        for (int i = 0; i < numModels; i++) {
            gather<precision>(XA[i], X, lVars,
                              hNVars[i], M, N,
                              i, streams[(i & (numStreams - 1))], *(new dim3(optimalBlock1D(hNVars[i] * M))));
        }
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            hipblasSetStream(hnd, streams[i & (numStreams - 1)]);
            gemm(hnd, HIPBLAS_OP_N, HIPBLAS_OP_T,
                 hNVars[i], hNVars[i], M,
                 alp[i], XA[i], hNVars[i],
                 XA[i], hNVars[i], bet[i],
                 G[i], hNVars[i]);
        }
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            hipblasSetStream(hnd, streams[i & (numStreams - 1)]);
            gemv(hnd, HIPBLAS_OP_N,
                 hNVars[i], M,
                 alp[i], XA[i], hNVars[i],
                 y + i * M, 1, bet[i],
                 r + i * M, 1);
        }
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            hipblasSetStream(hnd, streams[i & (numStreams - 1)]);
            getrfBatched(hnd, hNVars[i], dG + i,
                         hNVars[i], pivot + i * M, info + i, 1);
        }
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            hipblasSetStream(hnd, streams[i & (numStreams - 1)]);
            getriBatched(hnd, hNVars[i], dG + i,
                         hNVars[i], pivot + i * M, dI + i,
                         hNVars[i], info + i, 1);
        }
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            hipblasSetStream(hnd, streams[i & (numStreams - 1)]);
            gemv(hnd, HIPBLAS_OP_N,
                 hNVars[i], hNVars[i],
                 alp[i], I[i], hNVars[i],
                 r + i * M, 1, bet[i],
                 betaOls + i * M, 1);
        }
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            hipblasSetStream(hnd, streams[i & (numStreams - 1)]);    
            gemv(hnd, HIPBLAS_OP_T,
                 hNVars[i], M,
                 alp[i], XA[i], hNVars[i],
                 betaOls + i * M, 1, bet[i],
                 d + i * M, 1);
        }
        hipDeviceSynchronize();
        mat_sub<precision>(d, mu, d,
                           numModels * M, *(new dim3(bModM)));
        hipDeviceSynchronize();
        gammat<precision>(gamma, beta, betaOls, r,
                          lVars, nVars, M,
                          N, numModels, *(new dim3(bModM)));
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            hipblasSetStream(hnd, streams[i & (numStreams - 1)]);
            int length = (hNVars[i] > 1)? hNVars[i] - 1: 1;
            amin(hnd,
                 length, gamma + i * M,
                 1, dropidx + i);
        }
        hipDeviceSynchronize();
        hipblasSetStream(hnd, NULL);
        gemm(hnd, HIPBLAS_OP_N, HIPBLAS_OP_N,
             N, numModels, M,
             alp[0], X, N,
             d, M, bet[0],
             cd, N);
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            minCd(c + i * N, cd + i * N, cmax + i, r + i, N, streams[i & (numStreams - 1)], *(new dim3(1024)));
        }
        hipDeviceSynchronize();
        set_gamma<precision>(gamma, r, dropidx,
                             lasso, nVars, maxVariables,
                             M, numModels, *(new dim3(bMod)));
        hipDeviceSynchronize();
        update<precision>(beta, mu, d, cmax, r,
                          betaOls, gamma, lVars,
                          nVars, M, N,
                          numModels, *(new dim3(bModM)));
        hipDeviceSynchronize();
        drop(lVars, dropidx, nVars,
             lasso, M, numModels);
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            norm2(y + i * M, mu + i * M, r + i,
                  M, streams[i & (numStreams - 1)], *(new dim3(512)));
        }
        hipDeviceSynchronize();
        for (int i = 0; i < numModels; i++) {
            norm1(beta + i * N, cmax + i, N,
                  streams[i & (numStreams - 1)], *(new dim3(1024)));
        }
        hipDeviceSynchronize();
        final<precision>(a1, a2, cmax, r, step, done, numModels, 0.43, *(new dim3(bMod)));
        hipDeviceSynchronize();
        timer.stop();
        totalTime += timer.elapsed();
        totalFlop += 4.0 * (double) numModels * (double) N * (double) M;
        totalFlop += 7.0 * (double) numModels * (double) N;
        totalFlop += 6.0 * (double) numModels * (double) M;
        for (int i = 0; i < numModels; i++) {
            totalFlop += 2.0 * (double) hNVars[i] * (double) hNVars[i] * (double) M;
            totalFlop += (2.0 / 3.0) * (double) hNVars[i] * (double) hNVars[i] * (double) hNVars[i];
            totalFlop += 4.0 * (double) hNVars[i] * (double) M + 2.0 * (double) hNVars[i] * (double) hNVars[i];
            totalFlop += 5.0 * (double) hNVars[i];
        }
    }
    printf("\n");

    hipblasDestroy(hnd);
    hipFree(nVars);
    hipFree(step);
    hipFree(lasso);
    hipFree(done);
    hipFree(lVars);
    hipFree(cidx);
    hipFree(act);
    hipFree(dropidx);
    hipFree(pivot);
    hipFree(info);
    hipFree(ctrl);
    hipFree(hNVars);
    for (int i = 0; i < numModels; i++) {
        hipFree(alp[i]);
        hipFree(bet[i]);
        hipFree(XA[i]);
        hipFree(G[i]);
        hipFree(I[i]);
    }
    hipFree(y);
    hipFree(mu);
    hipFree(r);
    hipFree(beta);
    hipFree(c);
    hipFree(absC);
    hipFree(cmax);
    hipFree(betaOls);
    hipFree(d);
    hipFree(gamma);
    hipFree(cd);
    hipFree(a1);
    hipFree(a2);
    hipFree(dG);
    hipFree(dI);

    // for (int i = 0; i < totalModels; i++) {
    //     printf("Model = %d: a1 = %f: a2 = %f: nVars = %d\n", i, debug[i].a1, debug[i].a2, debug[i].nVars);
    // }
    printf("Gflops = %f\n", (totalFlop * 1.0e-9) / (totalTime * 1.0e-3));
    return 0;
}