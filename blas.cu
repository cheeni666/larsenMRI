#include "hip/hip_runtime.h"
#ifndef BLAS_CPP
#define BLAS_CPP

#include "blas.h"

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
          int m, int n, int k,
          const float *alpha, const float *A, int lda,
          const float *B, int ldb, const float *beta,
          float *C, int ldc) {
    hipblasSgemm(handle, transa, transb,
                m, n, k,
                alpha, A, lda,
                B, ldb, beta,
                C, ldc);
}

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
          int m, int n, int k,
          const double *alpha, const double *A, int lda,
          const double *B, int ldb, const double *beta,
          double *C, int ldc) {
    hipblasDgemm(handle, transa, transb,
                m, n, k,
                alpha, A, lda,
                B, ldb, beta,
                C, ldc);
}

void gemv(hipblasHandle_t handle, hipblasOperation_t trans,
          int m, int n,
          const float *alpha, const float *A, int lda,
          const float *x, int incx, const float *beta,
          float *y, int incy) {
    hipblasSgemv(handle, trans,
                m, n,
                alpha, A, lda,
                x, incx, beta,
                y, incy);
}

void gemv(hipblasHandle_t handle, hipblasOperation_t trans,
          int m, int n,
          const double *alpha, const double *A, int lda,
          const double *x, int incx, const double *beta,
          double *y, int incy) {
    hipblasDgemv(handle, trans,
                m, n,
                alpha, A, lda,
                x, incx, beta,
                y, incy);
}

void getrfBatched(hipblasHandle_t handle, int n, float *Aarray[],
                  int lda, int *PivotArray, int *infoArray,
                  int batchSize) {
    hipblasSgetrfBatched(handle, n, Aarray,
                        lda, PivotArray, infoArray,
                        batchSize);
}

void getrfBatched(hipblasHandle_t handle, int n, double *Aarray[],
                  int lda, int *PivotArray, int *infoArray,
                  int batchSize) {
    hipblasDgetrfBatched(handle, n, Aarray,
                        lda, PivotArray, infoArray,
                        batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, float *Aarray[],
                  int lda, int *PivotArray, float *Carray[],
                  int ldc, int *infoArray, int batchSize) {
    hipblasSgetriBatched(handle, n, (const float **)Aarray,
                       lda, PivotArray, Carray,
                       ldc, infoArray, batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, double *Aarray[],
                  int lda, int *PivotArray, double *Carray[],
                  int ldc, int *infoArray, int batchSize) {
    hipblasDgetriBatched(handle, n, (const double **)Aarray,
                       lda, PivotArray, Carray,
                       ldc, infoArray, batchSize);
}

void amax(hipblasHandle_t handle, int n, const float *x,
          int incx, int *result) {
    hipblasIsamax(handle, n, x,
                 incx, result);
}

void amax(hipblasHandle_t handle, int n, const double *x,
          int incx, int *result) {
    hipblasIdamax(handle, n, x,
                 incx, result);
}

void amin(hipblasHandle_t handle, int n, const float *x,
          int incx, int *result) {
    hipblasIsamin(handle, n, x,
                 incx, result);
}

void amin(hipblasHandle_t handle, int n, const double *x,
          int incx, int *result) {
    hipblasIdamin(handle, n, x,
                 incx, result);
}

//------------------------------------

template<class T>
struct SharedMemory {
    __device__ inline operator T *() {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<>
struct SharedMemory<double> {
    __device__ inline operator double *() {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

__device__
void SatomicMax(float *address, float val) {
    unsigned int* address_as_uint = (unsigned int*)address;
    unsigned int old = *address_as_uint, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_uint, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return;
}

__device__
void DatomicMax(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return;
}

__device__
void SatomicMin(float *address, float val) {
    unsigned int* address_as_uint = (unsigned int*)address;
    unsigned int old = *address_as_uint, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_uint, assumed, __float_as_int(fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return;
}

__device__
void DatomicMin(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return;
}

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__
void atomicAdd(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return;
}
#endif

__global__
void SamaxFabs_kernel(float *array, float *cmax, int elements) {
    float *smem = SharedMemory<float>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    smem[tid] = (gid < elements)? fabsf(array[gid]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < elements)
            smem[tid] = fmaxf(smem[tid], smem[tid + s]);
        __syncthreads();
    }
    if (tid == 0) {
        SatomicMax(cmax, smem[0]);
    }
}

__global__
void DamaxFabs_kernel(double *array, double *cmax, int elements) {
    double *smem = SharedMemory<double>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    smem[tid] = (gid < elements)? fabs(array[gid]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < elements)
            smem[tid] = fmax(smem[tid], smem[tid + s]);
        __syncthreads();
    }
    if (tid == 0) {
        DatomicMax(cmax, smem[0]);
    }
}

void amaxFabs(float *array, float *cmax, int elements, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((elements + blockDim.x - 1) / blockDim.x);
    SamaxFabs_kernel<<<gridDim, blockDim, blockDim.x * sizeof(float), stream>>>(array, cmax, elements);
}

void amaxFabs(double *array, double *cmax, int elements, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((elements + blockDim.x - 1) / blockDim.x);
    DamaxFabs_kernel<<<gridDim, blockDim, blockDim.x * sizeof(double), stream>>>(array, cmax, elements);
}

//------------------------------------

__global__
void SminCd_kernel(float *c, float *cd, float *cmax, float *r, int N) {
    float *smem = SharedMemory<float>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    float val = 50000;
    if (gid < N) {
        val = c[gid];
        if (val != 0) {
            float a = (val - *cmax) / (cd[gid] - *cmax);
            float b = (val + *cmax) / (cd[gid] + *cmax);
            a = (a <= 0)? 50000: a;
            b = (b <= 0)? 50000: b;
            val = min(a, b);
        }
        else {
            val = 50000;
        }
    }
    smem[tid] = val;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < N)
            smem[tid] = fminf(smem[tid], smem[tid + s]);
        __syncthreads();
    }
    if (tid == 0) {
        SatomicMin(r, smem[0]);
    }
}

__global__
void DminCd_kernel(double *c, double *cd, double *cmax, double *r, int N) {
    double *smem = SharedMemory<double>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    double val = 50000;
    if (gid < N) {
        val = c[gid];
        if (val != 0) {
            double a = (val - *cmax) / (cd[gid] - *cmax);
            double b = (val + *cmax) / (cd[gid] + *cmax);
            a = (a <= 0)? 50000: a;
            b = (b <= 0)? 50000: b;
            val = min(a, b);
        }
        else {
            val = 50000;
        }
    }
    smem[tid] = val;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < N)
            smem[tid] = fmin(smem[tid], smem[tid + s]);
        __syncthreads();
    }
    if (tid == 0) {
        DatomicMin(r, smem[0]);
    }
}

void minCd(float *c, float *cd, float *cmax, float *r, int N, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
    SminCd_kernel<<<gridDim, blockDim, blockDim.x * sizeof(float), stream>>>(c, cd, cmax, r, N);
}

void minCd(double *c, double *cd, double *cmax, double *r, int N, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
    DminCd_kernel<<<gridDim, blockDim, blockDim.x * sizeof(double), stream>>>(c, cd, cmax, r, N);
}

__global__
void Snorm2_kernel(float *y, float *mu, float *a2, int M) {
    float *smem = SharedMemory<float>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    smem[tid] = (gid < M)? (y[gid] - mu[gid]) * (y[gid] - mu[gid]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < M)
            smem[tid] += smem[tid + s];
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(a2, smem[0]);
    }
}

__global__
void Dnorm2_kernel(double *y, double *mu, double *a2, int M) {
    double *smem = SharedMemory<double>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    smem[tid] = (gid < M)? (y[gid] - mu[gid]) * (y[gid] - mu[gid]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < M)
            smem[tid] += smem[tid + s];
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(a2, smem[0]);
    }
}

void norm2(float *y, float *mu, float *a2, int M, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x);
    Snorm2_kernel<<<gridDim, blockDim, blockDim.x * sizeof(float), stream>>>(y, mu, a2, M);
}

void norm2(double *y, double *mu, double *a2, int M, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x);
    Dnorm2_kernel<<<gridDim, blockDim, blockDim.x * sizeof(double), stream>>>(y, mu, a2, M);
}

__global__
void Snorm1_kernel(float *beta, float *a1, int N) {
    float *smem = SharedMemory<float>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    smem[tid] = (gid < N)? fabsf(beta[gid]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < N)
            smem[tid] += smem[tid + s];
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(a1, smem[0]);
    }
}

__global__
void Dnorm1_kernel(double *beta, double *a1, int N) {
    double *smem = SharedMemory<double>();
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    smem[tid] = (gid < N)? fabs(beta[gid]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < N)
            smem[tid] += smem[tid + s];
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(a1, smem[0]);
    }
}

void norm1(float *beta, float *a1, int N, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
    Snorm1_kernel<<<gridDim, blockDim, blockDim.x * sizeof(float), stream>>>(beta, a1, N);
}

void norm1(double *beta, double *a1, int N, hipStream_t stream, dim3 blockDim) {
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
    Dnorm1_kernel<<<gridDim, blockDim, blockDim.x * sizeof(double), stream>>>(beta, a1, N);
}

#endif