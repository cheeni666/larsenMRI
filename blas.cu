#include "hip/hip_runtime.h"
#ifndef BLAS_CPP
#define BLAS_CPP

#include "blas.h"

int next_pow2(int num) {
	num--;
	num |= num >> 1;
	num |= num >> 2;
	num |= num >> 4;
	num |= num >> 8;
	num |= num >> 16;
	num++;
	return num;
}

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, float *alpha, float *A, int lda, float *B, int ldb, float *beta, float *C, int ldc) {
	hipblasSgemm(handle, transa, transb, m, n, k, (const float *)alpha, (const float *)A, lda, (const float *)B, ldb, (const float *)beta, C, ldc);
}

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, double *alpha, double *A, int lda, double *B, int ldb, double *beta, double *C, int ldc) {
	hipblasDgemm(handle, transa, transb, m, n, k, (const double *)alpha, (const double *)A, lda, (const double *)B, ldb, (const double *)beta, C, ldc);
}

void gemmBatched(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, float *alpha, float *Aarray[], int lda, float *Barray[], int ldb, float *beta, float *Carray[], int ldc, int batchCount) {
	hipblasSgemmBatched(handle, transa, transb, m, n, k, (const float *)alpha, (const float **)Aarray, lda, (const float **)Barray, ldb, (const float *)beta, Carray, ldc, batchCount);
}

void gemmBatched(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, double *alpha, double *Aarray[], int lda, double *Barray[], int ldb, double *beta, double *Carray[], int ldc, int batchCount) {
	hipblasDgemmBatched(handle, transa, transb, m, n, k, (const double *)alpha, (const double **)Aarray, lda, (const double **)Barray, ldb, (const double *)beta, Carray, ldc, batchCount);
}

void gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, float *alpha, float *A, int lda, float *x, int incx, float *beta, float *y, int incy) {
	hipblasSgemv(handle, trans, m, n, (const float *)alpha, (const float *)A, lda, (const float *)x, incx, (const float *)beta, y, incy);
}

void gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, double *alpha, double *A, int lda, double *x, int incx, double *beta, double *y, int incy) {
	hipblasDgemv(handle, trans, m, n, (const double *)alpha, (const double *)A, lda, (const double *)x, incx, (const double *)beta, y, incy);
}

void getrfBatched(hipblasHandle_t handle, int n, float *Aarray[], int lda, int *PivotArray, int *infoArray, int batchSize) {
	hipblasSgetrfBatched(handle, n, Aarray, lda, PivotArray, infoArray, batchSize);
}

void getrfBatched(hipblasHandle_t handle, int n, double *Aarray[], int lda, int *PivotArray, int *infoArray, int batchSize) {
	hipblasDgetrfBatched(handle, n, Aarray, lda, PivotArray, infoArray, batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, float *Aarray[], int lda, int *PivotArray, float *Carray[], int ldc, int *infoArray, int batchSize) {
	hipblasSgetriBatched(handle, n, (const float **)Aarray, lda, PivotArray, Carray, ldc, infoArray, batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, double *Aarray[], int lda, int *PivotArray, double *Carray[], int ldc, int *infoArray, int batchSize) {
	hipblasDgetriBatched(handle, n, (const double **)Aarray, lda, PivotArray, Carray, ldc, infoArray, batchSize);
}

void amax(hipblasHandle_t handle, int n, const float *x, int incx, int *result) {
	hipblasIsamax(handle, n, x, incx, result);
}

void amax(hipblasHandle_t handle, int n, const double *x, int incx, int *result) {
	hipblasIdamax(handle, n, x, incx, result);
}

void amin(hipblasHandle_t handle, int n, const float *x, int incx, int *result) {
	hipblasIsamin(handle, n, x, incx, result);
}

void amin(hipblasHandle_t handle, int n, const double *x, int incx, int *result) {
	hipblasIdamin(handle, n, x, incx, result);
}

//------------------------------------

template<class T>
struct SharedMemory {
	__device__ inline operator T *() {
		extern __shared__ int __smem[];
		return (T *)__smem;
	}

	__device__ inline operator const T *() const {
		extern __shared__ int __smem[];
		return (T *)__smem;
	}
};

template<>
struct SharedMemory<double> {
	__device__ inline operator double *() {
		extern __shared__ double __smem_d[];
		return (double *)__smem_d;
	}

	__device__ inline operator const double *() const {
		extern __shared__ double __smem_d[];
		return (double *)__smem_d;
	}
};

template<typename T>
__global__
void XAyBatched_kernel(T **XA, T *y, T *r, int *nVars, int M, int numModels) {
	int mod = threadIdx.x + blockIdx.x * blockDim.x;
	int ind = threadIdx.y + blockIdx.y * blockDim.y;
	if (mod < numModels) {
		int ni = nVars[mod];
		T *smem = SharedMemory<T>();
		smem[ind] = (ind < M)? y[mod * M + ind]: 0;
		__syncthreads();
		if (ind < ni) {
			T val = 0;
			for (int i = 0; i < M; i++) {
				val += XA[mod][ind * M + i] * smem[i];
			}
			r[mod * M + ind] = val;
		}
	}
}

template<typename T>
void XAyBatched(T **XA, T *y, T *r, int *nVars, int M, int numModels) {
	dim3 blockDim(1, M);
	dim3 gridDim(numModels, 1);
	XAyBatched_kernel<T><<<gridDim, blockDim, M * sizeof(T)>>>(XA, y, r, nVars, M, numModels);
}

template void XAyBatched<float>(float **XA, float *y, float *r, int *nVars, int M, int numModels);
template void XAyBatched<double>(double **XA, double *y, double *r, int *nVars, int M, int numModels);

template<typename T>
__global__
void IrBatched_kernel(T **I, T *r, T *betaOls, int *nVars, int M, int numModels) {
	int mod = threadIdx.x + blockIdx.x * blockDim.x;
	int ind = threadIdx.y + blockIdx.y * blockDim.y;
	if (mod < numModels) {
		int ni = nVars[mod];
		T *smem = SharedMemory<T>();
		smem[ind] = (ind < ni)? r[mod * M + ind]: 0;
		__syncthreads();
		if (ind < ni) {
			T val = 0;
			for (int i = 0; i < ni; i++) {
				val += I[mod][ind * ni + i] * smem[i];
			}
			betaOls[mod * M + ind] = val;
		}
	}
}

template<typename T>
void IrBatched(T **I, T *r, T *betaOls, int *nVars, int M, int numModels, int maxVar) {
	dim3 blockDim(1, maxVar);
	dim3 gridDim(numModels, 1);
	IrBatched_kernel<T><<<gridDim, blockDim, maxVar * sizeof(T)>>>(I, r, betaOls, nVars, M, numModels);
}

template void IrBatched<float>(float **I, float *r, float *betaOls, int *nVars, int M, int numModels, int maxVar);
template void IrBatched<double>(double **I, double *r, double *betaOls, int *nVars, int M, int numModels, int maxVar);

template<typename T>
__global__
void XAbetaOlsBatched_kernel(T **XA, T *betaOls, T *d, int *nVars, int M, int numModels) {
	int mod = threadIdx.x + blockIdx.x * blockDim.x;
	int ind = threadIdx.y + blockIdx.y * blockDim.y;
	if (mod < numModels) {
		int ni = nVars[mod];
		T *smem = SharedMemory<T>();
		if (ind < ni) smem[ind] = betaOls[mod * M + ind];
		__syncthreads();
		if (ind < M) {
			T val = 0;
			for (int i = 0; i < ni; i++) {
				val += XA[mod][i * M + ind] * smem[i];
			}
			d[mod * M + ind] = val;
		}
	}
}

template<typename T>
void XAbetaOlsBatched(T **XA, T *betaOls, T *d, int *nVars, int M, int numModels, int maxVar) {
	dim3 blockDim(1, M);
	dim3 gridDim(numModels, 1);
	XAbetaOlsBatched_kernel<T><<<gridDim, blockDim, maxVar * sizeof(T)>>>(XA, betaOls, d, nVars, M, numModels);
}

template void XAbetaOlsBatched<float>(float **XA, float *betaOls, float *d, int *nVars, int M, int numModels, int maxVar);
template void XAbetaOlsBatched<double>(double **XA, double *betaOls, double *d, int *nVars, int M, int numModels, int maxVar);

template<typename T>
__global__
void fabsMaxReduce_kernel(T *mat, T *buf, int *ind, int *intBuf, int rowSize, int colSize) {
	T *smem = SharedMemory<T>();
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	int tid = threadIdx.y;
	int col = threadIdx.y + blockIdx.y * blockDim.y;

	smem[tid] = (row < rowSize && col < colSize)? fabs(mat[row * colSize + col]): 0;
	if (ind == NULL) smem[tid + blockDim.y] = (row < rowSize && col < colSize)? col: 0;
	else smem[tid + blockDim.y] = (row < rowSize && col < colSize)? ind[row * colSize + col]: 0;
	__syncthreads();

	for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1) {
		if (tid < s && smem[tid + s] > smem[tid]) {
			smem[tid] = smem[tid + s];
			smem[tid + blockDim.y] = smem[tid + s + blockDim.y];
		}
		__syncthreads();
	}
	if (tid == 0) {
		buf[row * gridDim.y + blockIdx.y] = smem[0];
		intBuf[row * gridDim.y + blockIdx.y] = smem[blockDim.y];
	}
}

template<typename T>
void fabsMaxReduce(T *mat, T *res, T *buf, int *ind, int *intBuf, int rowSize, int colSize) {
	dim3 blockDim(1, 512);
	dim3 gridDim(rowSize, (colSize + blockDim.y - 1) / blockDim.y);
	fabsMaxReduce_kernel<T><<<gridDim, blockDim, 2 * blockDim.y * sizeof(T)>>>(mat, buf, NULL, intBuf, rowSize, colSize);
	colSize = gridDim.y;
	blockDim = *new dim3(1, next_pow2(colSize));
	gridDim = *new dim3(rowSize, 1);
	fabsMaxReduce_kernel<T><<<gridDim, blockDim, 2 * blockDim.y * sizeof(T)>>>(buf, res, intBuf, ind, rowSize, colSize);
}

template void fabsMaxReduce<float>(float *mat, float *res, float *buf, int *ind, int *intBuf, int rowSize, int colSize);
template void fabsMaxReduce<double>(double *mat, double *res, double *buf, int *ind, int *intBuf, int rowSize, int colSize);

template<typename T>
__global__
void cdMinReduce_kernel(T *c, T *cd, T *cmax, T *buf, int rowSize, int colSize, int opt) {
	T *smem = SharedMemory<T>();
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	int tid = threadIdx.y;
	int col = threadIdx.y + blockIdx.y * blockDim.y;

	smem[tid] = (row < rowSize && col < colSize)? c[row * colSize + col]: 50000;
	if (row < rowSize && col < colSize && opt) {
		if (smem[tid] == cmax[row]) smem[tid] = 0;
		if (smem[tid] != 0) {
			T a = (smem[tid] - cmax[row]) / (cd[row * colSize + col] - cmax[row]);
			T b = (smem[tid] + cmax[row]) / (cd[row * colSize + col] + cmax[row]);
			a = (a <= 0)? 50000: a;
			b = (b <= 0)? 50000: b;
			smem[tid] = min(a, b);
		}
		else {
			smem[tid] = 50000;
		}
	}
	__syncthreads();

	for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1) {
		if (tid < s && smem[tid + s] < smem[tid]) smem[tid] = smem[tid + s];
		__syncthreads();
	}
	if (tid == 0) {
		buf[row * gridDim.y + blockIdx.y] = smem[0];
	}
}

template<typename T>
void cdMinReduce(T *c, T *cd, T *cmax, T *res, T *buf, int rowSize, int colSize) {
	dim3 blockDim(1, 512);
	dim3 gridDim(rowSize, (colSize + blockDim.y - 1) / blockDim.y);
	cdMinReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(c, cd, cmax, buf, rowSize, colSize, 1);
	colSize = gridDim.y;
	blockDim = *new dim3(1, next_pow2(colSize));
	gridDim = *new dim3(rowSize, 1);
	cdMinReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(buf, NULL, NULL, res, rowSize, colSize, 0);
}

template void cdMinReduce<float>(float *c, float *cd, float *cmax, float *res, float *buf, int rowSize, int colSize);
template void cdMinReduce<double>(double *c, double *cd, double *cmax, double *res, double *buf, int rowSize, int colSize);

#endif
