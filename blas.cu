#include "hip/hip_runtime.h"
#ifndef BLAS_CPP
#define BLAS_CPP

#include "blas.h"

int next_pow2(int num) {
    num--;
    num |= num >> 1;
    num |= num >> 2;
    num |= num >> 4;
    num |= num >> 8;
    num |= num >> 16;
    num++;
    return num;
}

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
          int m, int n, int k,
          const float *alpha, const float *A, int lda,
          const float *B, int ldb, const float *beta,
          float *C, int ldc) {
    hipblasSgemm(handle, transa, transb,
                m, n, k,
                alpha, A, lda,
                B, ldb, beta,
                C, ldc);
}

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
          int m, int n, int k,
          const double *alpha, const double *A, int lda,
          const double *B, int ldb, const double *beta,
          double *C, int ldc) {
    hipblasDgemm(handle, transa, transb,
                m, n, k,
                alpha, A, lda,
                B, ldb, beta,
                C, ldc);
}

void gemv(hipblasHandle_t handle, hipblasOperation_t trans,
          int m, int n,
          const float *alpha, const float *A, int lda,
          const float *x, int incx, const float *beta,
          float *y, int incy) {
    hipblasSgemv(handle, trans,
                m, n,
                alpha, A, lda,
                x, incx, beta,
                y, incy);
}

void gemv(hipblasHandle_t handle, hipblasOperation_t trans,
          int m, int n,
          const double *alpha, const double *A, int lda,
          const double *x, int incx, const double *beta,
          double *y, int incy) {
    hipblasDgemv(handle, trans,
                m, n,
                alpha, A, lda,
                x, incx, beta,
                y, incy);
}

void getrfBatched(hipblasHandle_t handle, int n, float *Aarray[],
                  int lda, int *PivotArray, int *infoArray,
                  int batchSize) {
    hipblasSgetrfBatched(handle, n, Aarray,
                        lda, PivotArray, infoArray,
                        batchSize);
}

void getrfBatched(hipblasHandle_t handle, int n, double *Aarray[],
                  int lda, int *PivotArray, int *infoArray,
                  int batchSize) {
    hipblasDgetrfBatched(handle, n, Aarray,
                        lda, PivotArray, infoArray,
                        batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, float *Aarray[],
                  int lda, int *PivotArray, float *Carray[],
                  int ldc, int *infoArray, int batchSize) {
    hipblasSgetriBatched(handle, n, (const float **)Aarray,
                       lda, PivotArray, Carray,
                       ldc, infoArray, batchSize);
}

void getriBatched(hipblasHandle_t handle, int n, double *Aarray[],
                  int lda, int *PivotArray, double *Carray[],
                  int ldc, int *infoArray, int batchSize) {
    hipblasDgetriBatched(handle, n, (const double **)Aarray,
                       lda, PivotArray, Carray,
                       ldc, infoArray, batchSize);
}

void amax(hipblasHandle_t handle, int n, const float *x,
          int incx, int *result) {
    hipblasIsamax(handle, n, x,
                 incx, result);
}

void amax(hipblasHandle_t handle, int n, const double *x,
          int incx, int *result) {
    hipblasIdamax(handle, n, x,
                 incx, result);
}

void amin(hipblasHandle_t handle, int n, const float *x,
          int incx, int *result) {
    hipblasIsamin(handle, n, x,
                 incx, result);
}

void amin(hipblasHandle_t handle, int n, const double *x,
          int incx, int *result) {
    hipblasIdamin(handle, n, x,
                 incx, result);
}

//------------------------------------

template<class T>
struct SharedMemory {
    __device__ inline operator T *() {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<>
struct SharedMemory<double> {
    __device__ inline operator double *() {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

template<typename T>
__global__
void fabsMaxReduce_kernel(T *mat, T *buf, int rowSize, int colSize) {
    T *smem = SharedMemory<T>();
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.y;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    smem[tid] = (row < rowSize && col < colSize)? fabs(mat[row * colSize + col]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1) {
        if (tid < s)
            smem[tid] = max(smem[tid], smem[tid + s]);
        __syncthreads();
    }
    if (tid == 0) {
        buf[row * gridDim.y + blockIdx.y] = smem[0];
    }
}

template<typename T>
void fabsMaxReduce(T *mat, T *res, T *buf, int rowSize, int colSize) {
    dim3 blockDim(1, 1024);
    dim3 gridDim(rowSize, (colSize + blockDim.y - 1) / blockDim.y);
    fabsMaxReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(mat, buf, rowSize, colSize);
    colSize = gridDim.y;
    blockDim = *new dim3(1, next_pow2(colSize));
    gridDim = *new dim3(rowSize, 1);
    fabsMaxReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(buf, res, rowSize, colSize);
}

template void fabsMaxReduce<float>(float *mat, float *res, float *buf, int rowSize, int colSize);
template void fabsMaxReduce<double>(double *mat, double *res, double *buf, int rowSize, int colSize);

template<typename T>
__global__
void cdMinReduce_kernel(T *c, T *cd, T *cmax, T *buf, int rowSize, int colSize, int opt) {
    T *smem = SharedMemory<T>();
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.y;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    smem[tid] = (row < rowSize && col < colSize)? c[row * colSize + col]: 50000;
    if (row < rowSize && col < colSize && opt) {
        if (smem[tid] != 0) {
            T a = (smem[tid] - cmax[row]) / (cd[row * colSize + col] - cmax[row]);
            T b = (smem[tid] + cmax[row]) / (cd[row * colSize + col] + cmax[row]);
            a = (a <= 0)? 50000: a;
            b = (b <= 0)? 50000: b;
            smem[tid] = min(a, b);
        }
        else {
            smem[tid] = 50000;
        }
    }
    __syncthreads();

    for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1) {
        if (tid < s)
            smem[tid] = min(smem[tid], smem[tid + s]);
        __syncthreads();
    }
    if (tid == 0) {
        buf[row * gridDim.y + blockIdx.y] = smem[0];
    }
}

template<typename T>
void cdMinReduce(T *c, T *cd, T *cmax, T *res, T *buf, int rowSize, int colSize) {
    dim3 blockDim(1, 1024);
    dim3 gridDim(rowSize, (colSize + blockDim.y - 1) / blockDim.y);
    cdMinReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(c, cd, cmax, buf, rowSize, colSize, 1);
    colSize = gridDim.y;
    blockDim = *new dim3(1, next_pow2(colSize));
    gridDim = *new dim3(rowSize, 1);
    cdMinReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(buf, NULL, NULL, res, rowSize, colSize, 0);
}

template void cdMinReduce<float>(float *c, float *cd, float *cmax, float *res, float *buf, int rowSize, int colSize);
template void cdMinReduce<double>(double *c, double *cd, double *cmax, double *res, double *buf, int rowSize, int colSize);

template<typename T>
__global__
void fabsAddReduce_kernel(T *mat, T *buf, int rowSize, int colSize) {
    T *smem = SharedMemory<T>();
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.y;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    smem[tid] = (row < rowSize && col < colSize)? fabs(mat[row * colSize + col]): 0;
    __syncthreads();

    for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1) {
        if (tid < s)
            smem[tid] += smem[tid + s];
        __syncthreads();
    }
    if (tid == 0) {
        buf[row * gridDim.y + blockIdx.y] = smem[0];
    }
}

template<typename T>
void fabsAddReduce(T *mat, T *res, T *buf, int rowSize, int colSize) {
    dim3 blockDim(1, 1024);
    dim3 gridDim(rowSize, (colSize + blockDim.y - 1) / blockDim.y);
    fabsAddReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(mat, buf, rowSize, colSize);
    colSize = gridDim.y;
    blockDim = *new dim3(1, next_pow2(colSize));
    gridDim = *new dim3(rowSize, 1);
    fabsAddReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(buf, res, rowSize, colSize);
}

template void fabsAddReduce<float>(float *mat, float *res, float *buf, int rowSize, int colSize);
template void fabsAddReduce<double>(double *mat, double *res, double *buf, int rowSize, int colSize);

template<typename T>
__global__
void sqrAddReduce_kernel(T *y, T *mu, T *buf, int rowSize, int colSize, int opt) {
    T *smem = SharedMemory<T>();
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.y;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    smem[tid] = (row < rowSize && col < colSize)? y[row * colSize + col]: 0;
    if (row < rowSize && col < colSize && opt) {
        smem[tid] -= mu[row * colSize + col];
        smem[tid] *= smem[tid];
    }
    __syncthreads();

    for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1) {
        if (tid < s)
            smem[tid] += smem[tid + s];
        __syncthreads();
    }
    if (tid == 0) {
        buf[row * gridDim.y + blockIdx.y] = smem[0];
    }
}

template<typename T>
void sqrAddReduce(T *y, T *mu, T *res, T *buf, int rowSize, int colSize) {
    dim3 blockDim(1, 512);
    dim3 gridDim(rowSize, (colSize + blockDim.y - 1) / blockDim.y);
    sqrAddReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(y, mu, buf, rowSize, colSize, 1);
    colSize = gridDim.y;
    blockDim = *new dim3(1, next_pow2(colSize));
    gridDim = *new dim3(rowSize, 1);
    sqrAddReduce_kernel<T><<<gridDim, blockDim, blockDim.y * sizeof(T)>>>(buf, NULL, res, rowSize, colSize, 0);
}

template void sqrAddReduce<float>(float *y, float *mu, float *res, float *buf, int rowSize, int colSize);
template void sqrAddReduce<double>(double *y, double *mu, double *res, double *buf, int rowSize, int colSize);

#endif